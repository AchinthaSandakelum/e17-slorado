#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <ctime>
#include <iostream>
#include <hipblas.h>
#include <torch/torch.h>

#define BLOCK_THRESHOLD 1280000
#define TILE_X 32
#define TILE_Y 16
const float alpha = 1.0f; // Scaling factor for matrix A
const float beta = 1.0f;  // Scaling factor for matrix B
const float nbeta = -1.0f;
const float zbeta = 0.0f;
const dim3 blockDim(TILE_X * TILE_Y);
const dim3 gridDim((32/TILE_X) * (32/TILE_Y));

__global__ void computeC12 (float* C, float* m1 ,float* m2 ,float* m5 ,float* m6 , int width , int subWidth, int height, int subHeight )
{
    int tx = threadIdx.x; 
    /*This line obtains the x-coordinate (column) of the current thread within a thread block.*/
    int ty = threadIdx.y;
    /* This line obtains the y-coordinate (row) of the current thread within a thread block.*/
    int row = blockIdx.y * TILE_Y + ty;
    /* This line calculates the global row index in the original matrix for the current thread block using the blockIdx.y and threadIdx.y values. The variable 'TILE_Y' seems to represent the height of the tile.*/
    int column = blockIdx.x * TILE_X + tx;
    /* This line calculates the global column index in the original matrix for the current thread block using the blockIdx.x and threadIdx.x values. The variable 'TILE_X' seems to represent the width of the tile.*/
    __shared__ float as[ TILE_Y ][ TILE_X ];
    /* This line declares a shared memory array 'as' with dimensions TILE_Y by TILE_X. Shared memory is used for cooperative thread block-level data sharing.*/
    float Csub ; /*to store the intermediate sum.*/

    as[ty ][ tx ]= m1 [( row )* subWidth + column ];
    Csub =as[ty ][ tx ];
    as[ty ][ tx ]= m2 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m5 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m6 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    C[( row )* width + column ]= Csub;

}
__global__ void computeC11 (float* C,float* m2 ,float* m3,int width , int subWidth , int height, int subHeight)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_Y + ty;
    int column = blockIdx.x * TILE_X + tx;
    __shared__ float as[ TILE_Y ][ TILE_X ];
    float Csub;

    as[ty ][ tx ]= m2 [( row )* subWidth + column ];
    Csub =as[ty ][ tx ];
    as[ty ][ tx ]= m3 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    C[( row )* width + column ]= Csub;

}

__global__ void computeC21 (float* C,float* m1, float* m2, float* m4, float* m7 , int width , int subWidth, int height, int subHeight ){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_Y + ty;
    int column = blockIdx.x * TILE_X + tx;
    __shared__ float as[ TILE_Y ][ TILE_X ];
    float Csub ;

    as[ty ][ tx ]= m1 [( row )* subWidth + column ];
    Csub =as[ty ][ tx ];
    as[ty ][ tx ]= m2 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m4 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m7 [( row )* subWidth + column ];
    Csub -= as[ty ][ tx ];
    C[( row )* width + column ]= Csub;

}
__global__ void computeC22 (float* C,float* m1,float* m2, float* m4, float* m5 , int width , int subWidth, int height, int subHeight )
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * TILE_Y + ty;
    int column = blockIdx.x * TILE_X + tx;
    __shared__ float as[ TILE_Y ][ TILE_X ];
    float Csub ;

    as[ty ][ tx ]= m1 [( row )* subWidth + column ];
    Csub =as[ty ][ tx ];
    as[ty ][ tx ]= m2 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m4 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    as[ty ][ tx ]= m5 [( row )* subWidth + column ];
    Csub += as[ty ][ tx ];
    C[( row )* width + column ]= Csub;

}


__global__ void mergeSubmatrices(float* submatrix0, float* submatrix1, float* submatrix2, float* submatrix3, float* finalMatrix,int N, int M)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    
    finalMatrix[y * M + x] = submatrix0[y * N + x];
    finalMatrix[y * M + x + N] = submatrix1[y * N + x];
    finalMatrix[(y + N) * M + x] = submatrix2[y * N + x];
    finalMatrix[(y + N) * M + x + N] = submatrix3[y * N + x];
}


void winograd_mm(torch::Tensor const  &A,torch::Tensor const &B,torch::Tensor &C){

     auto start_time_mem_alloc = std::chrono::high_resolution_clock::now();
        //A, B, C are memory pointers on host
        // rowsB = colsA
        int rowsA = A.size(0);
        int colsA = A.size(1);
	int rowsB = B.size(0);
        int colsB = B.size(1);
        int subRowsA=rowsA/2;
        int subColsA=colsA/2;
        int subRowsB=rowsB/2;
        int subColsB=colsB/2;
     
     float *d_A11=0,*d_A12=0,*d_A21=0,*d_A22=0, *d_B11=0, *d_B12=0, *d_B21=0, *d_B22=0, *d_m1=0,*d_m2=0, *d_m3=0, *d_m4=0, *d_m5=0, *d_m6=0, *d_m7=0, *d_S1=0, *d_S2=0, *d_S3=0, *d_S4=0, *d_S5=0, *d_S6=0, *d_S7=0, *d_S8=0;
     float *d_C11=0, *d_C12=0, *d_C21=0, *d_C22=0, *d_C=0;
     hipMalloc((void**)&d_A11, subRowsA* subColsA * sizeof(float));
     hipMalloc((void**)&d_A12, subRowsA* subColsA * sizeof(float));
     hipMalloc((void**)&d_A21, subRowsA* subColsA * sizeof(float));
     hipMalloc((void**)&d_A22, subRowsA* subColsA * sizeof(float));

     hipMalloc((void**)&d_B11, subRowsB* subColsB * sizeof(float));
     hipMalloc((void**)&d_B12, subRowsB* subColsB * sizeof(float));
     hipMalloc((void**)&d_B21, subRowsB* subColsB * sizeof(float));
     hipMalloc((void**)&d_B22, subRowsB* subColsB * sizeof(float));


     auto start_time_split = std::chrono::high_resolution_clock::now();

     // Split the original matrix into four equal submatrices
     hipblasSetMatrix(subRowsA, subColsA, sizeof(float), (float *)A.data_ptr<float>(), colsA, d_A11, subColsA);
     hipblasSetMatrix(subRowsA, subColsA, sizeof(float), (float *)A.data_ptr<float>()+ subColsA, colsA, d_A12, subColsA);
     hipblasSetMatrix(subRowsA, subColsA, sizeof(float), (float *)A.data_ptr<float>()+ subRowsA * colsA, colsA, d_A21, subColsA);
     hipblasSetMatrix(subRowsA, subColsA, sizeof(float), (float *)A.data_ptr<float>()+ subRowsA * colsA+ subColsA,colsA, d_A22, subColsA);

     hipblasSetMatrix(subRowsB, subColsB, sizeof(float), (float *)B.data_ptr<float>(), colsB, d_B11, subColsB);
     hipblasSetMatrix(subRowsB, subColsB, sizeof(float), (float *)B.data_ptr<float>()+ subColsB, colsB, d_B12, subColsB);
     hipblasSetMatrix(subRowsB, subColsB, sizeof(float), (float *)B.data_ptr<float>()+ subRowsB * colsB, colsB, d_B21, subColsB);
     hipblasSetMatrix(subRowsB, subColsB, sizeof(float), (float *)B.data_ptr<float>()+ subRowsB * colsB+ subColsB, colsB, d_B22, subColsB);

     auto end_time_split = std::chrono::high_resolution_clock::now();
     hipblasHandle_t handle;
     hipblasStatus_t cudaStatus=hipblasCreate(&handle);

     hipMalloc((void**)&d_S1, subRowsA*subColsA*sizeof(float));
     hipMalloc((void**)&d_S2, subRowsA*subColsA*sizeof(float));
     hipMalloc((void**)&d_S3, subRowsA*subColsA*sizeof(float));
     hipMalloc((void**)&d_S4, subRowsA*subColsA*sizeof(float));
     hipMalloc((void**)&d_S5, subRowsB*subColsB*sizeof(float));
     hipMalloc((void**)&d_S6, subRowsB*subColsB*sizeof(float));
     hipMalloc((void**)&d_S7, subRowsB*subColsB*sizeof(float));
     hipMalloc((void**)&d_S8, subRowsB*subColsB*sizeof(float));

     hipMalloc((void**)&d_C11, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_C12, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_C21, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_C22, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_C, rowsA*colsB*sizeof(float));

     auto end_time_mem_alloc = std::chrono::high_resolution_clock::now();

     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsA, subColsA,
                          &alpha,
                          d_A21, subRowsA,
                          &beta,
                          d_A22, subRowsA,
                          d_S1, subRowsA);
     // add(A21,A22,S1);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsA, subColsA,
                          &alpha,
                          d_A11, subRowsA,
                          &nbeta,
                          d_A21, subRowsA,
                          d_S3, subRowsA);
     // add(A11,A21,S3);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsB, subColsB,
                          &alpha,
                          d_B12, subRowsB,
                          &nbeta,
                          d_B11, subRowsB,
                          d_S5, subRowsB);
     //sub(B12,B11,S5);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsB, subColsB,
                          &alpha,
                          d_B22, subRowsB,
                          &nbeta,
                          d_B12, subRowsB,
                          d_S7, subRowsB);
     // sub(B22,B12,S7);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsA, subColsA,
                          &alpha,
                          d_S1, subRowsA,
                          &nbeta,
                          d_A11, subRowsA,
                          d_S2, subRowsA);
     // sub(S1,A11,S2);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsB, subColsB,
                          &alpha,
                          d_B22, subRowsB,
                          &nbeta,
                          d_S5, subRowsB,
                          d_S6, subRowsB);
     // sub(B22,S5,S6);
     hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsA, subColsA,
                          &alpha,
                          d_A12, subRowsA,
                          &nbeta,
                          d_S2, subRowsA,
                          d_S4, subRowsA);
    // sub(A12,S2,S4);
    hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          subRowsB, subColsB,
                          &alpha,
                          d_S6, subRowsB,
                          &nbeta,
                          d_B21, subRowsB,
                          d_S8, subRowsB);

     hipMalloc((void**)&d_m1, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m2, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m3, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m4, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m5, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m6, subRowsA*subColsB*sizeof(float));
     hipMalloc((void**)&d_m7, subRowsA*subColsB*sizeof(float));

    auto end_time_sgeam = std::chrono::high_resolution_clock::now();
    auto start_time_sgemm=std::chrono::high_resolution_clock::now(), end_time_sgemm=std::chrono::high_resolution_clock::now(), start_time_recursive=std::chrono::high_resolution_clock::now() ,end_time_recursive=std::chrono::high_resolution_clock::now();
    if (rowsA <= BLOCK_THRESHOLD){
	start_time_sgemm = std::chrono::high_resolution_clock::now();
        //hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_S6, subColsB, d_S2, subColsA, &zbeta, d_m1, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_B11, subColsB, d_A11, subColsA, &zbeta, d_m2, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_B21, subColsB, d_A12, subColsA, &zbeta, d_m3, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_S7, subColsB, d_S3, subColsA, &zbeta, d_m4, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_S5, subColsB, d_S1, subColsA, &zbeta, d_m5, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_B22, subColsB, d_S4, subColsA, &zbeta, d_m6, subColsB);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, subColsB, subRowsA, subColsA, &alpha, d_S8, subColsB, d_A22, subColsA, &zbeta, d_m7, subColsB);
	end_time_sgemm = std::chrono::high_resolution_clock::now();
   } else {
	/*	
	//start_time_recursive = std::chrono::high_resolution_clock::now();
        winograd_mm(d_S2, d_S6, d_m1, subRowsA, subColsA, subRowsB, subColsB);
        winograd_mm(d_A11, d_B11, d_m2, subRowsA, subColsA,subRowsB,subColsB);
        winograd_mm(d_A12, d_B21, d_m3, subRowsA, subColsA,subRowsB,subColsB);
        winograd_mm(d_S3, d_S7, d_m4, subRowsA, subColsA,subRowsB,subColsB);
        winograd_mm(d_S1, d_S5, d_m5, subRowsA, subColsA,subRowsB,subColsB);
        winograd_mm(d_S4, d_B22, d_m6, subRowsA, subColsA,subRowsB,subColsB);
        winograd_mm(d_A22, d_S8, d_m7, subRowsA, subColsA,subRowsB,subColsB);
	//end_time_recursive = std::chrono::high_resolution_clock::now();
	*/
    }
    auto start_time_free = std::chrono::high_resolution_clock::now();
    hipDeviceSynchronize();
    hipFree(d_A11);
    hipFree(d_A12);
    hipFree(d_A21);
    hipFree(d_A22);
    hipFree(d_B11);
    hipFree(d_B12);
    hipFree(d_B21);
    hipFree(d_B22);
    hipFree(d_S1);
    hipFree(d_S2);
    hipFree(d_S3);
    hipFree(d_S4);
    hipFree(d_S5);
    hipFree(d_S6);
    hipFree(d_S7);
    hipFree(d_S8);

    auto start_time_kernels = std::chrono::high_resolution_clock::now();

    computeC11<<<gridDim,blockDim>>> (d_C11, d_m2, d_m3, colsB, subColsB, rowsA, subRowsA);
    computeC12<<<gridDim,blockDim>>> (d_C12, d_m1, d_m2 , d_m5 , d_m6, colsB, subColsB, rowsA, subRowsA);
    computeC21<<<gridDim,blockDim>>> (d_C21, d_m1 , d_m2 , d_m4, d_m7, colsB, subColsB, rowsA, subRowsA);
    computeC22<<<gridDim,blockDim>>> (d_C22, d_m1, d_m2, d_m4, d_m5 , colsB, subColsB, rowsA, subRowsA);

    auto end_time_kernels = std::chrono::high_resolution_clock::now();

    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);
    hipFree(d_m4);
    hipFree(d_m5);
    hipFree(d_m6);
    hipFree(d_m7);

    auto start_time_merge = std::chrono::high_resolution_clock::now();
    // Launch the CUDA kernel to merge submatrices
    dim3 threadsPerBlock(subRowsA, subRowsA);
    mergeSubmatrices<<<16,threadsPerBlock>>>(d_C11, d_C12, d_C21, d_C22, d_C, subRowsA, rowsA);

    //hipMemcpy(C, d_C, rowsA * colsB * sizeof(float), hipMemcpyDeviceToHost);
    //convert d_C to deviceTensor
    torch::Tensor deviceTensor = torch::from_blob(d_C, {rowsA,colsB});

    C= deviceTensor.to(torch::kCPU);
    auto end_time_merge = std::chrono::high_resolution_clock::now();

    auto elapsed_time_mem_alloc = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_mem_alloc  - end_time_split)+std::chrono::duration_cast<std::chrono::milliseconds>(start_time_split - start_time_mem_alloc);
    auto elapsed_time_split = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_split  - start_time_split);
    auto elapsed_time_sgeam = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_sgeam - end_time_mem_alloc);
    auto elapsed_time_sgemm = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_sgemm - start_time_sgemm);
    //auto elapsed_time_recursive = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_recursive   - start_time_recursive );
    auto elapsed_time_kernels = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_kernels - start_time_kernels );
    auto elapsed_time_merge = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_merge - start_time_merge);
    auto elapsed_time_free = std::chrono::duration_cast<std::chrono::milliseconds>(start_time_merge - end_time_kernels)+std::chrono::duration_cast<std::chrono::milliseconds>(start_time_kernels  - start_time_free);

    std::cout << "Elapsed time mem alloc: " << elapsed_time_mem_alloc.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time split: " << elapsed_time_split.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time sgeam: " << elapsed_time_sgeam.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time sgemm: " << elapsed_time_sgemm.count() << " milliseconds" << std::endl;
    //std::cout << "Elapsed time recursive: " << elapsed_time_recursive.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time kernels: " << elapsed_time_kernels.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time merge: " << elapsed_time_merge.count() << " milliseconds" << std::endl;
    std::cout << "Elapsed time free: " << elapsed_time_free.count() << " milliseconds" << std::endl;

}
